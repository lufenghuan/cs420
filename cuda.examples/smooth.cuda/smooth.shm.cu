/*******************************************************************************
*
*  smooth.shm.cu
*
*  CUDA shared memory version.
*
*  This provides a CUDA implementation of a kernel smooother.
*   http://en.wikipedia.org/wiki/Kernel_smoother
*  The particular smoother in this file is a nearest neighbor smoother
*  in order to keep the code as simple to understand as possible.
*
*  This is implemeneted for 2-d square grids.
*
*  Parameters of note:
*    dataWidth -- size of the data is dataWidth^2
*    halfWidth -- region around point x,y to smooth
*        k smooths box with corners [x-k,y-k] to [x+k,y+k]
*
*  The smoothed region is only defined for the interior that has the kernel
*   defined inside the boundary, e.g. for dataWidth=10, halfWidth=2 the
*   region from 2,2 to 7,7 will be smoothed. 
*
********************************************************************************/

/*******************************************************************************
*
*  CUDA concepts
*
*  This file shows how to use many features of CUDA:
*     2d grids
*     pitch allocation
*     shared memory
*
********************************************************************************/

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <assert.h>
#include <sys/time.h>

#include <hip/hip_runtime.h>

// Data is of size dataWidth * dataWidth
//const unsigned int dataWidth = 4112;

// Parameter to express the smoothing kernel halfwidth
//const unsigned int halfWidth = 8;

// Size of the CUDA threadBlock
//const unsigned int blockWidth = 16;

/* Small values good for testing */

// Data is of size dataWidth * dataWidth
const unsigned int dataWidth = 8;

// Parameter to express the smoothing kernel halfwidth
const unsigned int halfWidth = 1;

// Size of the CUDA threadBlock
const unsigned int blockWidth = 2;



/*------------------------------------------------------------------------------
* Name: NNSmoothKernel
* Action:  The CUDA kernel that implements kernel smoothing.
*             Yuck, that's two senses of kernel.
*-----------------------------------------------------------------------------*/
__global__ void NNSmoothKernel ( float* pFieldIn, float* pFieldOut, size_t pitch )
{ 
  extern __shared__ float shared[][blockWidth+2*halfWidth];

  // pitch is in bytes, figure out the number of elements for addressing
  unsigned pitchels = pitch/sizeof(float);

  // Each node loads one element at it's threadIdx
  shared[threadIdx.x][threadIdx.y] = 
    pFieldIn [  (blockIdx.y * blockDim.y + threadIdx.y) * pitchels 
                   +  blockIdx.x * blockDim.x + threadIdx.x ];

  // Load the right portion beyond the threadBlock
  if ( threadIdx.x < 2*halfWidth )
  {
    shared[threadIdx.x + blockWidth][threadIdx.y] = 
      pFieldIn [  (blockIdx.y * blockDim.y + threadIdx.y) * pitchels 
                     +  blockIdx.x * blockDim.x + threadIdx.x + blockWidth ];
  }

  // Load the bottom portion beyond the threadBlock
  if ( threadIdx.y < 2*halfWidth )
  {
    shared[threadIdx.x][threadIdx.y + blockWidth] = 
      pFieldIn [  (blockIdx.y * blockDim.y + threadIdx.y + blockWidth) * pitchels 
                     +  blockIdx.x * blockDim.x + threadIdx.x];
  }

  // Load the bottom right portion beyond the threadBlock
  if ( ( threadIdx.y < 2*halfWidth ) && ( threadIdx.x < 2*halfWidth ))
  {
    shared[threadIdx.x + blockWidth][threadIdx.y + blockWidth] = 
      pFieldIn [  (blockIdx.y * blockDim.y + threadIdx.y + blockWidth) * pitchels 
                     +  blockIdx.x * blockDim.x + threadIdx.x + blockWidth];
  }

  __syncthreads();


  // Variable to accumulate the smoothed value
  float value = 0.0;

  // The grid indexes start from 
  unsigned xindex = ( blockIdx.x * blockDim.x + threadIdx.x) + halfWidth; 
  unsigned yindex = ( blockIdx.y * blockDim.y + threadIdx.y) + halfWidth; 

  // Get the value from the kernel
  for ( unsigned j=0; j<2*halfWidth+1; j++ )
  {
    for ( unsigned i=0; i<2*halfWidth+1; i++ )
    {
      value += shared [threadIdx.x+i] [threadIdx.y+j];
    }
  }
  
  // Divide by the number of elements in the kernel
  value /= (2*halfWidth+1)*(2*halfWidth+1);

  // Write the value out 
  pFieldOut [ yindex*pitchels + xindex ] = value;

} 


/*------------------------------------------------------------------------------
* Name:  SmoothField
* Action:  Host entry point to kernel smoother
*-----------------------------------------------------------------------------*/
bool SmoothField ( float* pHostFieldIn, float *pHostFieldOut ) 
{ 
  float * pDeviceFieldIn = 0;
  float * pDeviceFieldOut = 0;

  size_t pitch, pitchout;

  struct timeval ta, tb, tc, td;

  // Check the grid dimensions and extract parameters.  See top description about restrictions
  assert((dataWidth-(2*halfWidth)) % blockWidth == 0 );

  gettimeofday ( &ta, NULL );

  // Place the data set on device memory
  hipMallocPitch((void**)&pDeviceFieldIn, &pitch, dataWidth*sizeof(float), dataWidth ); 
  hipMemcpy2D ( pDeviceFieldIn, pitch,
                 pHostFieldIn, dataWidth*sizeof(float), dataWidth*sizeof(float), dataWidth,
                 hipMemcpyHostToDevice); 

  // Allocate the output
  hipMallocPitch((void**)&pDeviceFieldOut, &pitchout, dataWidth*sizeof(float), dataWidth ); 

  gettimeofday ( &tb, NULL );

  // Construct a 2d grid/block
  const dim3 DimBlock ( blockWidth, blockWidth );
  const dim3 DimGrid ( (dataWidth-(2*halfWidth))/blockWidth , 
                       (dataWidth-(2*halfWidth))/blockWidth );
  const unsigned shmemSize = ( blockWidth + 2*halfWidth) * ( blockWidth + 2*halfWidth ) * sizeof (float);

  // Invoke the kernel
  NNSmoothKernel <<<DimGrid,DimBlock, shmemSize>>> ( pDeviceFieldIn, pDeviceFieldOut, pitch ); 

  gettimeofday ( &tc, NULL );

  // Retrieve the results
  hipMemcpy2D(pHostFieldOut, dataWidth*sizeof(float), 
               pDeviceFieldOut, pitch, dataWidth*sizeof(float), dataWidth,
               hipMemcpyDeviceToHost); 

  gettimeofday ( &td, NULL );


  if ( ta.tv_usec < td.tv_usec )
  {
    printf ("Elapsed total time (s/m): %ld:%d\n", td.tv_sec - ta.tv_sec, td.tv_usec - ta.tv_usec );
  } else {
    printf ("Elapsed total time (s/m): %ld:%d\n", td.tv_sec - ta.tv_sec - 1, 1000000 - td.tv_usec + ta.tv_usec );
  }

  if ( tb.tv_usec < tc.tv_usec )
  {
    printf ("Elapsed kernel time (s/m): %ld:%d\n", tc.tv_sec - tb.tv_sec, tc.tv_usec - tb.tv_usec );
  } else {
    printf ("Elapsed kernel time (s/m): %ld:%d\n", tc.tv_sec - tb.tv_sec - 1, 1000000 - tc.tv_usec + tb.tv_usec );
  }

  return true;
}



/*------------------------------------------------------------------------------
* Name:  initField
* Action:  Initialize a field to predictable values.
*    This is a useful format for debugging, because values 
*    accumulate to their initial value.
*-----------------------------------------------------------------------------*/
void initField ( unsigned dim, float* pField )
{
  for ( unsigned j=0; j<dim; j++ )
  {
    for ( unsigned i=0; i<dim; i++ )
    {
      pField[j*dim+i] = j + i;
    }
  }
}


/*------------------------------------------------------------------------------
* Name:  main
* Action:  Entry point
*-----------------------------------------------------------------------------*/
int main ()
{
  // Create the input field
  float *field = (float *) malloc ( dataWidth * dataWidth * sizeof(float));
  initField ( dataWidth, field );

  // Create the output field
  float *out = (float *) malloc ( dataWidth * dataWidth * sizeof(float));

  // Call the kernel
  SmoothField ( field, out );

  // Print the output field (for debugging purposes.
  unsigned koffset = halfWidth;
  for ( unsigned j=0; j< dataWidth; j++ )
  {
    for ( unsigned i=0; i< dataWidth; i++ )
    {
      if ( ( i >= koffset ) && 
           ( j >= koffset ) &&
           ( i < ( dataWidth - koffset )) &&
           ( j < ( dataWidth - koffset )) )
      {
        printf ("%4.4f, ", out[j*dataWidth + i]);
      }
      else
      {
        printf ("%4.4f, ", 0.0f );
      }
    }  
    printf ("\n");
  }

}
